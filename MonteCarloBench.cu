#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define IL_BLOKOW 256
#define IL_WATKOW 256
#define IL_WEWN_TESTOW 1024

#define PI 3.14159265358979323846 // przyblizenie liczby pi do 20 miejsc po przecinku

// Cudowna wersja metody Monte Carlo
__global__ void cuda_monte_carlo(float *wyniki, hiprandState *stany) 
{
	unsigned int moje_id = threadIdx.x + blockDim.x * blockIdx.x;
	int i;
	long k = 0;
	float x, y;

	hiprand_init(moje_id*moje_id, moje_id, 0, &stany[moje_id]); // Inicjalizacja CURAND

	for(i = 0; i < IL_WEWN_TESTOW; i++)
	{
		x = hiprand_uniform(&stany[moje_id]);
		y = hiprand_uniform(&stany[moje_id]);
		if((x * x + y * y) <= 1.0f) k++;
	}

	wyniki[moje_id] = (4 * (float)k / IL_WEWN_TESTOW);
}

// Sekwencyjna wersja na procesorze
float proc_sekw_monte_carlo(long ilosc_testow) 
{
	long k, i;
	float x, y;
	srand(time(NULL));

	for(i = 0; i < ilosc_testow; i++) 
	{
		x = rand() / (float) RAND_MAX;
		y = rand() / (float) RAND_MAX;
		if((x * x + y * y) <= 1.0f) k++;
	}

	return (4 * (float)k / ilosc_testow);
}

// Funkcja main
int main(int argc, char *argv[])
{
	int i;
	clock_t start, stop;
	float *gfx_w;
	hiprandState *gfx_stany;
	float proc[IL_BLOKOW * IL_WATKOW];
	
	printf("\t-> Ilosc blokow: %d.\n\t-> Ilosc watkow na blok: %d.\n\t-> Ilosc testow dla kazdego watku: %d.\n\n", IL_BLOKOW, IL_WATKOW, IL_WEWN_TESTOW);
	
	/*****************************************
        * Start wersji na CUDA
        *****************************************/
	
	start = clock();
	
	hipMalloc((void **)&gfx_w, IL_WATKOW * IL_BLOKOW * sizeof(float));
	hipMalloc((void **)&gfx_stany, IL_WATKOW * IL_BLOKOW * sizeof(hiprandState));
	
	cuda_monte_carlo<<<IL_BLOKOW, IL_WATKOW>>>(gfx_w, gfx_stany);
	hipMemcpy(proc, gfx_w, IL_WATKOW * IL_BLOKOW * sizeof(float), hipMemcpyDeviceToHost);
	
	float pi_cuda;
	
	for(i = 0; i < IL_WATKOW * IL_BLOKOW; i++)
	{
		pi_cuda += proc[i];
	}
	
	pi_cuda /= (IL_WATKOW * IL_BLOKOW);
	
	stop = clock();
	
	printf("\t-> Czas liczenia PI na CUDA: %.6f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);
	printf("\t-> Wartosc PI wg CUDA: %.10f (blad wzgledem rzeczywistej wartosci: %.10f).\n", pi_cuda, pi_cuda - PI);

	// ----------- Koniec wersji na CUDA
	
	/*****************************************
        * Start wersji sekwencyjnej na procesorze
        *****************************************/
        
	start = clock();
	
	float pi_proc_sekw = proc_sekw_monte_carlo(IL_WATKOW * IL_BLOKOW * IL_WEWN_TESTOW);
	
	stop = clock();
	
	printf("\t-> Czas liczenia PI sekwencyjnie na procesorze: %.6f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);
	printf("\t-> Wartosc PI wg CPU (sekw.): %.10f (blad wzgledem rzeczywistej wartosci: %.10f).\n", pi_proc_sekw, pi_proc_sekw - PI);

	// ----------- Koniec wersji sekwensyjnej na procesorze
	
	return 0;
}
